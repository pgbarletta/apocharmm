#include "hip/hip_runtime.h"
#include<pybind11/pybind11.h>
#include<pybind11/stl.h>

#include "Coordinates.h"
#include "CharmmCrd.h"
#include "PDB.h"

namespace py = pybind11;

void exportCoordinates(py::module & mod){
  py::class_<Coordinates, std::shared_ptr<Coordinates>>(mod, "Coordinates")
    //.def(py::init<const std::string &>(), "Handle for charmm .crd file")
    .def("getNumAtoms", &CharmmCrd::getNumAtoms, R"pbdoc(
      :return: The number of atoms
      :rtype: int
      )pbdoc")
    .def("getCoordinates", &CharmmCrd::getCoordinates, R"sitb(
      :return: coordinates of all atoms
      :rtype: vector<float4>
      )sitb");

}
void exportCharmmCrd(py::module & mod){
  py::class_<CharmmCrd, std::shared_ptr<CharmmCrd>, Coordinates>(mod, "CharmmCrd")
    .def(py::init<const std::string &>(), "Handle for charmm .crd file")
    //.def("getNumAtoms", &CharmmCrd::getNumAtoms, "number of atoms")
    //.def("getCoordinates", &CharmmCrd::getCoordinates, "coordinates of all atoms");
    ;
}

void exportPDB(py::module & mod){
  py::class_<PDB, std::shared_ptr<PDB>, Coordinates>(mod, "PDB", R"sitb(
     
     PDB file content

     Can be used to extract coordinate data via getCoordinates.
     
     )sitb")
    .def(py::init<const std::string &>(), "Handle for .pdb file")
    //.def("getNumAtoms", &CharmmCrd::getNumAtoms, "number of atoms")
    //.def("getCoordinates", &CharmmCrd::getCoordinates, "coordinates of all atoms");
    ;
}

