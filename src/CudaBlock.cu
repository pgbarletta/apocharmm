// BEGINLICENSE
//
// This file is part of chcuda, which is distributed under the BSD 3-clause
// license, as described in the LICENSE file in the top level directory of this
// project.
//
// Author: Antti-Pekka Hynninen, Samarjeet Prasad
//
// ENDLICENSE

#ifndef NOCUDAC
#include "CudaBlock.h"
#include "cuda_utils.h"
#include "gpu_utils.h"
#include <cassert>
#include <iostream>
#ifndef USE_TEXTURE_OBJECTS
#include "CudaDirectForceKernels.h"
#endif

//#ifndef USE_TEXTURE_OBJECTS
// VdW parameter texture reference
// texture<float, 1, hipReadModeElementType> blockParamTexRef;
//#endif

//
// Class creator
//
CudaBlock::CudaBlock(const int numBlock, const int use_softcore,
                     const int use_PMEL)
    : numBlock(numBlock) {
  assert(numBlock >= 1);
  blockTypeLen = 0;
  blockType = NULL;
  useSoftcore = use_softcore;
  usePMEL = use_PMEL;
  allocate<float>(&d_blockParam, numBlock * (numBlock + 1) / 2);
  allocate_host<float>(&h_blockParam, numBlock * (numBlock + 1) / 2);
  if (usePMEL >= 2) {
    allocate<float>(&d_blockParamEx, numBlock * (numBlock + 1) / 2);
    allocate_host<float>(&h_blockParamEx, numBlock * (numBlock + 1) / 2);
  }
  if (useSoftcore == 2) {
    allocate<double>(&d_DSoftDFscale, numBlock * (numBlock + 1) / 2);
  }
  allocate<float>(&bixlam, numBlock);
  allocate<double>(&biflam, numBlock);
  allocate<double>(&biflam2, numBlock);
  allocate<int>(&siteMLD, numBlock);
#ifdef USE_TEXTURE_OBJECTS
  // Use texture objects
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = d_blockParam;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = sizeof(float) * 8;
  resDesc.res.linear.sizeInBytes =
      numBlock * (numBlock + 1) / 2 * sizeof(float);
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  cudaCheck(
      hipCreateTextureObject(&blockParamTexObj, &resDesc, &texDesc, NULL));
#else
  assert(!getBlockParamTexRefBound());
  // Bind blockparam texture
  memset(getBlockParamTexRef(), 0,
         sizeof(texture<float, 1, hipReadModeElementType>));
  getBlockParamTexRef()->normalized = 0;
  getBlockParamTexRef()->filterMode = hipFilterModePoint;
  getBlockParamTexRef()->addressMode[0] = hipAddressModeClamp;
  getBlockParamTexRef()->channelDesc.x = 32;
  getBlockParamTexRef()->channelDesc.y = 0;
  getBlockParamTexRef()->channelDesc.z = 0;
  getBlockParamTexRef()->channelDesc.w = 0;
  getBlockParamTexRef()->channelDesc.f = hipChannelFormatKindFloat;
  cudaCheck(hipBindTexture(NULL, *getBlockParamTexRef(), d_blockParam,
                            numBlock * (numBlock + 1) / 2 * sizeof(float)));
  setBlockParamTexRefBound(true);
#endif
}

//
// Class destructor
//
CudaBlock::~CudaBlock() {
#ifdef USE_TEXTURE_OBJECTS
  cudaCheck(hipDestroyTextureObject(blockParamTexObj));
#else
  cudaCheck(hipUnbindTexture(*getBlockParamTexRef()));
  setBlockParamTexRefBound(false);
#endif
  if (blockType != NULL)
    deallocate<int>(&blockType);
  deallocate<float>(&d_blockParam);
  deallocate_host<float>(&h_blockParam);
  if (usePMEL >= 2) {
    deallocate<float>(&d_blockParamEx);
    deallocate_host<float>(&h_blockParamEx);
  }
  if (useSoftcore == 2) {
    deallocate<double>(&d_DSoftDFscale);
  }
  deallocate<float>(&bixlam);
  deallocate<double>(&biflam);
  deallocate<double>(&biflam2);
  deallocate<int>(&siteMLD);
}

//
// Sets blocktype array from host memory
//
void CudaBlock::setBlockType(const int ncoord, const int *h_blockType) {
  // Align ncoord to warpsize
  int ncoord_aligned = ((ncoord - 1) / warpsize + 1) * warpsize;
  reallocate<int>(&blockType, &blockTypeLen, ncoord_aligned, 1.2f);
  copy_HtoD_sync<int>(h_blockType, blockType, ncoord);
}

//
// Sets block parameters by copying them from CPU
// NOTE: The CPU buffer is in full-matrix form
//
void CudaBlock::setBlockParam(const float *h_blockParamFull) {
  int k = 0;
  for (int i = 0; i < numBlock; i++) {
    for (int j = 0; j <= i; j++) {
      h_blockParam[k] = h_blockParamFull[j * numBlock + i];
      if (usePMEL == 2) {
        h_blockParamEx[k] = h_blockParamFull[j * numBlock + i];
        if (i == j) {
          h_blockParamEx[k] *= h_blockParamFull[j * numBlock + i];
        }
      } else if (usePMEL == 3) {
        h_blockParamEx[k] = h_blockParamFull[i] * h_blockParamFull[j];
      }
      k++;
    }
  }
  copy_HtoD_sync<float>(h_blockParam, d_blockParam,
                        numBlock * (numBlock + 1) / 2);
  if (usePMEL >= 2) {
    copy_HtoD_sync<float>(h_blockParamEx, d_blockParamEx,
                          numBlock * (numBlock + 1) / 2);
  }
}

//
// Sets bixlam by copying them from CPU
//
void CudaBlock::setBixlam(const float *h_bixlam) {
  copy_HtoD_sync<float>(h_bixlam, bixlam, numBlock);
}

//
// Set siteMLD
//
void CudaBlock::setSiteMLD(const int *h_siteMLD) {
  copy_HtoD_sync<int>(h_siteMLD, siteMLD, numBlock);
}

//
// Copies biflam and biflam2 to CPU arrays
//
void CudaBlock::getBiflam(double *h_biflam, double *h_biflam2) {
  copy_DtoH_sync<double>((double *)biflam, h_biflam, numBlock);
  copy_DtoH_sync<double>((double *)biflam2, h_biflam2, numBlock);
}

//
// Sets biflam and biflam2 GPU arrays
//
void CudaBlock::setBiflam(double *h_biflam, double *h_biflam2) {
  copy_HtoD_sync<double>(h_biflam, (double *)biflam, numBlock);
  copy_HtoD_sync<double>(h_biflam2, (double *)biflam2, numBlock);
}
#endif // NOCUDAC
