// BEGINLICENSE
//
// This file is part of chcuda, which is distributed under the BSD 3-clause
// license, as described in the LICENSE file in the top level directory of this
// project.
//
// Author: Antti-Pekka Hynninen, Samarjeet Prasad
//
// ENDLICENSE

#include <CudaIntegratorGraph.h>
CudaIntegratorGraph::CudaIntegratorGraph() {
  cudaCheck(hipGraphCreate(&graph, 0));
  hipGraphNode_t emptynode;
  cudaCheck(hipGraphAddEmptyNode(&emptynode, graph, NULL, 0));
}
hipGraph_t CudaIntegratorGraph::getGraph() { return graph; }
CudaIntegratorGraph::~CudaIntegratorGraph() {
  cudaCheck(hipGraphDestroy(graph));
}
