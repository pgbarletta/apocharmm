#include "hip/hip_runtime.h"
// BEGINLICENSE
//
// This file is part of chcuda, which is distributed under the BSD 3-clause
// license, as described in the LICENSE file in the top level directory of this
// project.
//
// Author: Antti-Pekka Hynninen, Samarjeet Prasad
//
// ENDLICENSE

#ifndef NOCUDAC
#include "XYZQ.h"
#include "cuda_utils.h"
#include "gpu_utils.h"
#include <cassert>
#include <fstream>
#include <iostream>
#include <sstream>

//
// XYZQ class method definitions
//
// (c) Antti-Pekka Hynninen, 2013, aphynninen@hotmail.com
//
//

//
// Copies x, y, z coordinates into xyzq -array
//
__global__ void set_xyz_kernel(const int ncoord, const double *__restrict__ x,
                               const double *__restrict__ y,
                               const double *__restrict__ z,
                               float4 *__restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < ncoord) {
    xyzq[tid].x = x[tid];
    xyzq[tid].y = y[tid];
    xyzq[tid].z = z[tid];
  }
}

//
// Copies (x, y, z, q) into xyzq -array
//
__global__ void set_xyzq_kernel(const int ncoord, const double *__restrict__ x,
                                const double *__restrict__ y,
                                const double *__restrict__ z,
                                const float *__restrict__ q,
                                float4 *__restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    xyzq_val.x = x[tid];
    xyzq_val.y = y[tid];
    xyzq_val.z = z[tid];
    xyzq_val.w = q[tid];
    xyzq[tid] = xyzq_val;
  }
}

//
// Copies (x, y, z, q) into xyzq -array and also shifts (x, y, z)
//
__global__ void set_xyzq_shift_kernel(
    const int ncoord, const double *__restrict__ x,
    const double *__restrict__ y, const double *__restrict__ z,
    const float *__restrict__ q, const int *__restrict__ loc2glo,
    const float3 *__restrict__ xyz_shift, const double boxx, const double boxy,
    const double boxz, float4 *__restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    float3 shift = xyz_shift[tid];
    xyzq_val.x = (float)(x[tid] + ((double)shift.x) * boxx);
    xyzq_val.y = (float)(y[tid] + ((double)shift.y) * boxy);
    xyzq_val.z = (float)(z[tid] + ((double)shift.z) * boxz);
    xyzq_val.w = q[loc2glo[tid]];
    xyzq[tid] = xyzq_val;
  }
}

//
// Copies (x, y, z) into xyzq -array and also shifts (x, y, z)
//
__global__ void
set_xyz_shift_kernel(const int ncoord, const double *__restrict__ x,
                     const double *__restrict__ y, const double *__restrict__ z,
                     const float3 *__restrict__ xyz_shift, const double boxx,
                     const double boxy, const double boxz,
                     float4 *__restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    float3 shift = xyz_shift[tid];
    xyzq_val.x = (float)(x[tid] + ((double)shift.x) * boxx);
    xyzq_val.y = (float)(y[tid] + ((double)shift.y) * boxy);
    xyzq_val.z = (float)(z[tid] + ((double)shift.z) * boxz);
    xyzq[tid].x = xyzq_val.x;
    xyzq[tid].y = xyzq_val.y;
    xyzq[tid].z = xyzq_val.z;
  }
}

//##########################################################################################
//##########################################################################################
//##########################################################################################

//
// Return xyzq length that has extra align:
// ncoord-1 = last possible index
//
int XYZQ::get_xyzq_len(const int ncoord_in) {
  return ((ncoord_in - 1) / align + 1) * align;
}

__global__ void clear(int size, float4 *xyzq) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < size) {
    xyzq[index].x = 0.0f;
    xyzq[index].y = 0.0f;
    xyzq[index].z = 0.0f;
    xyzq[index].w = 0.0f;
  }
}

void XYZQ::set_ncoord(const int ncrd) {
  align = warpsize;
  ncoord = ncrd;
  xyzq_len = get_xyzq_len(ncoord);
  allocate<float4>(&xyzq, xyzq_len);

  int numThreads = 128;
  int numBlocks = (ncoord - 1) / numThreads + 1;
  clear<<<numBlocks, numThreads>>>(ncoord, xyzq);
  hipDeviceSynchronize();

  host_xyzq =
      std::make_shared<std::vector<float4>>(std::vector<float4>(ncoord));
  // printf("Size of host array is : %d ", host_xyzq->size());
  // host_xyzq->resize(xyzq_len);
}

//
// Class creator
//
XYZQ::XYZQ() {
  ncoord = 0;
  xyzq_len = 0;
  align = warpsize;
  xyzq = NULL;
}

//
// Class creator
//
XYZQ::XYZQ(int ncoord, int align) : ncoord(ncoord), align(align) {
  xyzq_len = get_xyzq_len(ncoord);
  allocate<float4>(&xyzq, xyzq_len);
  host_xyzq->resize(xyzq_len);
}

//
// Class creator
//
XYZQ::XYZQ(const char *filename, int align) : align(align) {
  std::ifstream file(filename);
  if (file.is_open()) {
    float x, y, z, q;

    // Count number of coordinates
    ncoord = 0;
    while (file >> x >> y >> z >> q)
      ncoord++;

    // Rewind
    file.clear();
    file.seekg(0, std::ios::beg);

    // Allocate CPU memory
    float4 *xyzq_cpu = new float4[ncoord];

    // Read coordinates
    int i = 0;
    while (file >> xyzq_cpu[i].x >> xyzq_cpu[i].y >> xyzq_cpu[i].z >>
           xyzq_cpu[i].w)
      i++;

    // Allocate GPU memory
    xyzq_len = get_xyzq_len(ncoord);
    allocate<float4>(&xyzq, xyzq_len);

    // Copy coordinates from CPU to GPU
    copy_HtoD<float4>(xyzq_cpu, xyzq, ncoord);

    // Deallocate CPU memory
    delete[] xyzq_cpu;

  } else {
     std::stringstream tmpexc; 
    tmpexc << "Error opening file " << filename << std::endl;
    throw std::invalid_argument(tmpexc.str());
    exit(1);
  }
}

//
// Class destructor
//
XYZQ::~XYZQ() {
  if (xyzq != NULL)
    deallocate<float4>(&xyzq);
}

//
// Re-allocates array, does not preserve content
//
void XYZQ::realloc(int ncoord_new, float fac) {
  reallocate<float4>(&xyzq, &xyzq_len, get_xyzq_len(ncoord_new), fac);
  this->ncoord = ncoord_new;
}

//
// Re-sizes array, preserves content
//
void XYZQ::resize(int ncoord_new, float fac) {
  ::resize<float4>(&xyzq, &xyzq_len, ncoord, get_xyzq_len(ncoord_new), fac);
  this->ncoord = ncoord_new;
}

//
// Copies xyzq from host
// NOTE: Does not reallocate xyzq
//
void XYZQ::set_xyzq(int ncopy, const float4 *h_xyzq, size_t offset,
                    hipStream_t stream) {
  copy_HtoD<float4>(&h_xyzq[offset], &xyzq[offset], ncopy, stream);
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyzq(const cudaXYZ<double> &coord, const float *q,
                    hipStream_t stream) {
  assert(ncoord == coord.size());

  int nthread = 512;
  int nblock = (ncoord - 1) / nthread + 1;

  set_xyzq_kernel<<<nblock, nthread, 0, stream>>>(
      coord.size(), coord.x(), coord.y(), coord.z(), q, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyzq(const cudaXYZ<double> &coord, const float *q,
                    const int *loc2glo, const float3 *xyz_shift,
                    const double boxx, const double boxy, const double boxz,
                    hipStream_t stream) {
  assert(ncoord == coord.size());

  int nthread = 512;
  int nblock = (ncoord - 1) / nthread + 1;

  set_xyzq_shift_kernel<<<nblock, nthread, 0, stream>>>(
      coord.size(), coord.x(), coord.y(), coord.z(), q, loc2glo, xyz_shift,
      boxx, boxy, boxz, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z (on device) into the coordinate slots
//
void XYZQ::set_xyz(const cudaXYZ<double> &coord, hipStream_t stream) {
  assert(ncoord == coord.size());

  int nthread = 512;
  int nblock = (ncoord - 1) / nthread + 1;

  set_xyz_kernel<<<nblock, nthread, 0, stream>>>(coord.size(), coord.x(),
                                                 coord.y(), coord.z(), xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyz(const cudaXYZ<double> &coord, const int start, const int end,
                   const float3 *xyz_shift, const double boxx,
                   const double boxy, const double boxz, hipStream_t stream) {
  assert(ncoord == coord.size());
  assert(start >= 0);
  assert(end < ncoord);
  assert(end < coord.size());
  int nset = end - start + 1;
  assert(nset >= 0);

  if (nset == 0)
    return;

  int nthread = 512;
  int nblock = (nset - 1) / nthread + 1;

  set_xyz_shift_kernel<<<nblock, nthread, 0, stream>>>(
      nset, coord.x() + start, coord.y() + start, coord.z() + start,
      &xyz_shift[start], boxx, boxy, boxz, &xyzq[start]);

  cudaCheck(hipGetLastError());
}

//
// Compares two XYZQ arrays
//
bool XYZQ::compare(XYZQ &xyzq_in, const double tol, double &max_diff) {
  assert(xyzq_in.ncoord == ncoord);

  float4 *h_xyzq = new float4[ncoord];
  float4 *h_xyzq_in = new float4[ncoord];
  copy_DtoH<float4>(xyzq, h_xyzq, ncoord);
  copy_DtoH<float4>(xyzq_in.xyzq, h_xyzq_in, ncoord);

  bool ok = true;

  max_diff = 0.0;
  int i;
  double dx, dy, dz, dq;
  double diff;
  try {
    for (i = 0; i < ncoord; i++) {
      dx = fabs(h_xyzq[i].x - h_xyzq_in[i].x);
      dy = fabs(h_xyzq[i].y - h_xyzq_in[i].y);
      dz = fabs(h_xyzq[i].z - h_xyzq_in[i].z);
      dq = fabs(h_xyzq[i].w - h_xyzq_in[i].w);
      diff = max(dx, max(dy, dz));
      max_diff = max(max_diff, diff);
      if (diff > tol || dq > 0.0)
        throw 1;
    }
  } catch (int a) {
    std::cout << "i = " << i << std::endl;
    std::cout << "this: x,y,z,q = " << h_xyzq[i].x << " " << h_xyzq[i].y << " "
              << h_xyzq[i].z << " " << h_xyzq[i].w << std::endl;
    std::cout << "in  : x,y,z,q = " << h_xyzq_in[i].x << " " << h_xyzq_in[i].y
              << " " << h_xyzq_in[i].z << " " << h_xyzq_in[i].w << std::endl;
    ok = false;
  }

  delete[] h_xyzq;
  delete[] h_xyzq_in;

  return ok;
}

//
// Print to ostream
//
void XYZQ::print(const int start, const int end, std::ostream &out) {
  float4 *h_xyzq = new float4[ncoord];
  copy_DtoH_sync<float4>(xyzq, h_xyzq, ncoord);

  for (int i = start; i <= end; i++) {
    out << i << " " << h_xyzq[i].x << " " << h_xyzq[i].y << " " << h_xyzq[i].z
        << " " << h_xyzq[i].w << std::endl;
  }

  delete[] h_xyzq;
}

//
// Save to file
//
void XYZQ::save(const char *filename) {
  std::ofstream file(filename);
  if (file.is_open()) {
    float4 *h_xyzq = new float4[ncoord];
    copy_DtoH_sync<float4>(xyzq, h_xyzq, ncoord);
    for (int i = 0; i < ncoord; i++) {
      file << h_xyzq[i].x << " " << h_xyzq[i].y << " " << h_xyzq[i].z << " "
           << h_xyzq[i].w << std::endl;
    }
    delete[] h_xyzq;
  } else {
     std::stringstream tmpexc; 
    tmpexc << "Error opening file " << filename << std::endl;
    throw std::invalid_argument(tmpexc.str());
    exit(1);
  }
}

float4 *XYZQ::getDeviceXYZQ() { return xyzq; }

void XYZQ::transferFromDevice() {
  hipMemcpy(host_xyzq->data(), xyzq, sizeof(float4) * host_xyzq->size(),
             hipMemcpyDeviceToHost);
}

void XYZQ::transferToDevice() {
  hipMemcpy(xyzq, host_xyzq->data(), sizeof(float4) * host_xyzq->size(),
             hipMemcpyHostToDevice);
}

std::shared_ptr<std::vector<float4>> XYZQ::getHostXYZQ() {
  transferFromDevice();
  return host_xyzq;
}

void XYZQ::setDeviceXYZQ(float4 *in) { xyzq = in; }

void XYZQ::set_xyz(const std::vector<float> &coords) {
  transferFromDevice();
  for (int i = 0; i < ncoord; i++) {
    host_xyzq->at(i).x = coords[i * 3];
    host_xyzq->at(i).y = coords[i * 3 + 1];
    host_xyzq->at(i).z = coords[i * 3 + 2];
  }
  transferToDevice();
}

std::vector<float> XYZQ::get_xyz() {
  transferFromDevice();
  std::vector<float> coords(ncoord * 3);
  for (int i = 0; i < ncoord; i++) {
    coords[i * 3] = host_xyzq->at(i).x;
    coords[i * 3 + 1] = host_xyzq->at(i).y;
    coords[i * 3 + 2] = host_xyzq->at(i).z;
  }
  return coords;
}

std::vector<float> XYZQ::get_q() {
   transferFromDevice();
   std::vector<float> qs(ncoord);
   for (int i = 0; i < ncoord; i++) {
      qs[i] = host_xyzq->at(i).w;
   }
   return qs;
}

__global__ static void imageCenterKernel(float3 boxSize, int numGroups,
                                         const int2 *__restrict__ groups,
                                         float4 *__restrict__ xyzq) {

  int index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index < numGroups) {
    int2 group = groups[index];

    float gx = 0.0;
    float gy = 0.0;
    float gz = 0.0;
    for (int i = group.x; i <= group.y; ++i) {
      gx += xyzq[i].x;
      gy += xyzq[i].y;
      gz += xyzq[i].z;
    }
    gx /= (group.y - group.x + 1);
    gy /= (group.y - group.x + 1);
    gz /= (group.y - group.x + 1);

    if (gx < -0.5 * boxSize.x) {
      for (int i = group.x; i <= group.y; ++i) {
        xyzq[i].x += boxSize.x;
      }
    }

    if (gx > 0.5 * boxSize.x) {
      for (int i = group.x; i <= group.y; ++i) {
        xyzq[i].x -= boxSize.x;
      }
    }

    if (gy < -0.5 * boxSize.y) {
      for (int i = group.x; i <= group.y; ++i) {
        xyzq[i].y += boxSize.y;
      }
    }

    if (gy > 0.5 * boxSize.y) {
      for (int i = group.x; i <= group.y; ++i) {
        xyzq[i].y -= boxSize.y;
      }
    }

    if (gz < -0.5 * boxSize.z) {
      for (int i = group.x; i <= group.y; ++i) {
        xyzq[i].z += boxSize.z;
      }
    }

    if (gz > 0.5 * boxSize.z) {
      for (int i = group.x; i <= group.y; ++i) {
        xyzq[i].z -= boxSize.z;
      }
    }
  }
}

void XYZQ::imageCenter(const std::vector<float> &boxDimensions,
                       // CudaContainer<int4> &groups) {
                       CudaContainer<int2> &groups) {

  int numGroups = groups.size();
  int numThreads = 128;
  int numBlocks = (numGroups - 1) / numThreads + 1;

  float3 boxSize = {boxDimensions[0], boxDimensions[1], boxDimensions[2]};
  // auto dv = groups.getDeviceArray().data();
  imageCenterKernel<<<numBlocks, numThreads>>>(
      boxSize, numGroups, groups.getDeviceArray().data(), xyzq);
  cudaCheck(hipDeviceSynchronize());
}

void XYZQ::orient(bool massWeighting, bool rotation) {}

#endif // NOCUDAC
