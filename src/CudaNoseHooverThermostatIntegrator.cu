#include "hip/hip_runtime.h"
// BEGINLICENSE
//
// This file is part of chcuda, which is distributed under the BSD 3-clause
// license, as described in the LICENSE file in the top level directory of this
// project.
//
// Author: Antti-Pekka Hynninen, Samarjeet Prasad, James E. Gonzales II
//
// ENDLICENSE

#include "Constants.h"
#include "CudaNoseHooverThermostatIntegrator.h"
#include "gpu_utils.h"
#include <chrono>
#include <iostream>

CudaNoseHooverThermostatIntegrator::CudaNoseHooverThermostatIntegrator(
    const double timeStep)
    : CudaIntegrator(timeStep) {
  m_ChainLength = 5;
  m_StepId = 0;
  m_NoseHooverPistonMass = 0.0;
  m_NoseHooverPistonPosition = 0.0;
  m_NoseHooverPistonVelocity = 0.0;
  m_NoseHooverPistonVelocityPrevious = 0.0;
  m_NoseHooverPistonForce = 0.0;
  m_NoseHooverPistonForcePrevious = 0.0;
  m_BathTemperature = 300.0;

  m_IntegratorTypeName = "CudaNoseHooverThermostatIntegrator";
}

__global__ static void init(double kbt, const int numAtoms, const int stride,
                            const double timeStep,
                            // double4 *__restrict__ coords,
                            double4 *__restrict__ coordsDelta,
                            double4 *__restrict__ coordsDeltaPrevious,
                            const double4 *__restrict__ velMass,
                            const double *__restrict__ force) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < numAtoms) {
    double fx = -force[index];
    double fy = -force[index + stride];
    double fz = -force[index + 2 * stride];

    double fact = timeStep * timeStep * velMass[index].w * 0.5;

    coordsDeltaPrevious[index].x = velMass[index].x * timeStep - fx * fact;
    coordsDeltaPrevious[index].y = velMass[index].y * timeStep - fy * fact;
    coordsDeltaPrevious[index].z = velMass[index].z * timeStep - fz * fact;

    coordsDelta[index].x = velMass[index].x * timeStep + fx * fact;
    coordsDelta[index].y = velMass[index].y * timeStep + fy * fact;
    coordsDelta[index].z = velMass[index].z * timeStep + fz * fact;
  }
}

__global__ static void
backStepInitializationKernel(int numAtoms, double4 *__restrict__ coords,
                             double4 *__restrict__ coordsDeltaPrevious) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < numAtoms) {
    coords[index].x -= coordsDeltaPrevious[index].x;
    coords[index].y -= coordsDeltaPrevious[index].y;
    coords[index].z -= coordsDeltaPrevious[index].z;
  }
}

__global__ static void
backStepInitializationKernel2(int numAtoms, double4 *__restrict__ coords,
                              double4 *__restrict__ coordsRef,
                              double4 *__restrict__ coordsDeltaPrevious) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < numAtoms) {
    coordsDeltaPrevious[index].x = coordsRef[index].x - coords[index].x;
    coordsDeltaPrevious[index].y = coordsRef[index].y - coords[index].y;
    coordsDeltaPrevious[index].z = coordsRef[index].z - coords[index].z;

    coords[index].x = coordsRef[index].x;
    coords[index].y = coordsRef[index].y;
    coords[index].z = coordsRef[index].z;
  }
}

static __global__ void
updateSPKernel(int numAtoms, float4 *__restrict__ xyzq,
               const double4 *__restrict__ coordsCharge) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < numAtoms) {
    xyzq[index].x = (float)coordsCharge[index].x;
    xyzq[index].y = (float)coordsCharge[index].y;
    xyzq[index].z = (float)coordsCharge[index].z;
  }
}

void CudaNoseHooverThermostatIntegrator::initialize(void) {
  int numAtoms = m_Context->getNumAtoms();

  m_NoseHooverPistonMass = 500.0; // TODO : set this
  m_NoseHooverPistonVelocity = 0.0;
  m_NoseHooverPistonVelocityPrevious = 0.0;
  m_NoseHooverPistonForce = 0.0;
  m_NoseHooverPistonForcePrevious = 0.0;

  m_CoordsDelta.resize(numAtoms);
  m_CoordsDeltaPrevious.resize(numAtoms);

  auto coordsRefDevice = m_CoordsRef.getDeviceData();
  // if (m_UsingHolonomicConstraints) {
  //   m_HolonomicConstraintForces.resize(numAtoms);
  // }

  int numThreads = 128;
  int numBlocks = (numAtoms - 1) / numThreads + 1;

  auto coords = m_Context->getCoordinatesCharges().getDeviceData();
  auto xyzq = m_Context->getXYZQ()->getDeviceXYZQ();
  auto coordsDeltaDevice = m_CoordsDelta.getDeviceData();
  auto coordsDeltaPreviousDevice = m_CoordsDeltaPrevious.getDeviceData();
  auto velMass = m_Context->getVelocityMass().getDeviceData();

  if (m_UsingHolonomicConstraints) {
    copy_DtoD_async<double4>(coords, coordsRefDevice, numAtoms,
                             *m_IntegratorStream);
    cudaCheck(hipStreamSynchronize(*m_IntegratorStream));
    // cudaCheck(hipDeviceSynchronize());

    m_HolonomicConstraint->handleHolonomicConstraints(coordsRefDevice);
    updateSPKernel<<<numBlocks, numThreads, 0, *m_IntegratorStream>>>(
        numAtoms, xyzq, coords);
    copy_DtoD_async<double4>(coords, coordsRefDevice, numAtoms,
                             *m_IntegratorStream);
    cudaCheck(hipStreamSynchronize(*m_IntegratorStream));
    // cudaCheck(hipDeviceSynchronize());
  }

  m_Context->calculateForces();
  auto force = m_Context->getForces();

  int stride = m_Context->getForceStride();
  double kbt = charmm::constants::kBoltz * m_BathTemperature;

  init<<<numBlocks, numThreads, 0, *m_IntegratorStream>>>(
      kbt, numAtoms, stride, m_TimeStep, // coords,
      coordsDeltaDevice, coordsDeltaPreviousDevice, velMass, force->xyz());
  cudaCheck(hipStreamSynchronize(*m_IntegratorStream));
  // cudaCheck(hipDeviceSynchronize());

  if (m_UsingHolonomicConstraints) {
    backStepInitializationKernel<<<numBlocks, numThreads, 0,
                                   *m_IntegratorStream>>>(
        numAtoms, coords, coordsDeltaPreviousDevice);

    m_HolonomicConstraint->handleHolonomicConstraints(coordsRefDevice);

    backStepInitializationKernel2<<<numBlocks, numThreads, 0,
                                    *m_IntegratorStream>>>(
        numAtoms, coords, coordsRefDevice, coordsDeltaPreviousDevice);
  }
  cudaCheck(hipStreamSynchronize(*m_IntegratorStream));

  m_StepId = 0;

  return;
}

// change this to save delta

// static __global__ void firstHalfKickAndDrift(const int numAtoms,
//                                              const int stride,
//                                              const ts_t timeStep, float4
//                                              *xyzq, double4 *velMass, const
//                                              double *__restrict__ force) {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if (index < numAtoms) {
//     float fx = force[index];
//     float fy = force[index + stride];
//     float fz = force[index + 2 * stride];
//
//     velMass[index].x -= 0.5 * timeStep * fx * velMass[index].w;
//     velMass[index].y -= 0.5 * timeStep * fy * velMass[index].w;
//     velMass[index].z -= 0.5 * timeStep * fz * velMass[index].w;
//
//     xyzq[index].x += timeStep * velMass[index].x;
//     xyzq[index].y += timeStep * velMass[index].y;
//     xyzq[index].z += timeStep * velMass[index].z;
//   }
// }
//
// static __global__ void secondHalfKick(const int numAtoms, const int stride,
//                                       const ts_t timeStep, float4 *xyzq,
//                                       double4 *velMass, const double *force)
//                                       {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if (index < numAtoms) {
//
//     float fx = force[index];
//     float fy = force[index + stride];
//     float fz = force[index + 2 * stride];
//
//     velMass[index].x -= 0.5 * timeStep * fx * velMass[index].w;
//     velMass[index].y -= 0.5 * timeStep * fy * velMass[index].w;
//     velMass[index].z -= 0.5 * timeStep * fz * velMass[index].w;
//   }
// }
//
//__global__ void ke(const double4 *velMass, int numAtoms, double *d_ke) {
//   extern __shared__ double sdata[];
//
//   unsigned int id = threadIdx.x;
//   unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
//
//   if (tid < numAtoms) {
//     sdata[id] =
//         (velMass[tid].x * velMass[tid].x + velMass[tid].y * velMass[tid].y +
//          velMass[tid].z * velMass[tid].z) /
//         velMass[tid].w;
//   } else {
//     sdata[id] = 0.0f;
//   }
//   __syncthreads();
//
//   for (int s = 1; s < blockDim.x; s *= 2) {
//     if (id % (s * 2) == 0) {
//       sdata[id] += sdata[id + s];
//     }
//     __syncthreads();
//
//     if (id == 0) {
//       d_ke[blockIdx.x] = (double)sdata[0];
//     }
//   }
// }
//
/*
 Velocity-Verlet drift step propagator
*/
// static __global__ void u1Propagator(float deltaT, int numAtoms,
//                                     const double4 *__restrict__ velMass,
//                                     double4 *__restrict__ xyzq) {

//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if (index < numAtoms) {
//     xyzq[index].x += deltaT * velMass[index].x;
//     xyzq[index].y += deltaT * velMass[index].y;
//     xyzq[index].z += deltaT * velMass[index].z;
//   }
// }

// /*
// Velocty-Verlet kick step propagator
// */
// static __global__ void u2Propagator(float deltaT, int numAtoms, int stride,
//                                     double4 *__restrict__ velMass,
//                                     const double *__restrict__ force,
//                                     const double4 *__restrict__ xyzq) {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if (index < numAtoms) {

//     const double fx = force[index];
//     const double fy = force[index + stride];
//     const double fz = force[index + 2 * stride];

//     velMass[index].x -= deltaT * fx * velMass[index].w;
//     velMass[index].y -= deltaT * fy * velMass[index].w;
//     velMass[index].z -= deltaT * fz * velMass[index].w;
//   }
// }

///*
// Thermostat propagator
// chain positions
//*/
//  __global__ void
// u3Propagator(float deltaT, int numAtoms, double4 *__restrict__ velMass,
//              int chainLength, double *__restrict__ chainPositions,
//              const double *__restrict__ chainVelocities) {
//   unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
//
//   if (index < numAtoms) {
//     velMass[index].x *= exp(-deltaT * chainVelocities[0]);
//     velMass[index].y *= exp(-deltaT * chainVelocities[0]);
//     velMass[index].z *= exp(-deltaT * chainVelocities[0]);
//   }
//
//   if (index == 0) {
//     for (int i = 0; i < chainLength; i++) {
//       chainPositions[i] += deltaT * chainVelocities[i];
//     }
//   }
// }
//
///*
// Thermostat propagator
// chain velocities are updated
//*/
//__global__ void u434Propagator(float deltaT, int numAtoms, double kT,
//                                       int ndegf, double4 *__restrict__
//                                       velMass, int chainLength, double
//                                       *__restrict__ chainPositions, double
//                                       *__restrict__ chainVelocities, const
//                                       double *__restrict__ chainMasses) {
//
//   // Calculate p * p / m
//   unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if (index < numAtoms) {
//   }
//
//   double kineticEnergy = 0.0; // TODO : calculate it
//   int M = chainLength - 1;
//   double G =
//       ((chainVelocities[M - 1] * chainVelocities[M - 1]) * chainMasses[M - 1]
//       -
//        kT) /
//       chainMasses[M];
//
//   chainVelocities[M] += deltaT * G;
//
//   for (int m = M - 1; m >= 0; m--) {
//     G = ((chainVelocities[m - 1] * chainVelocities[m - 1]) *
//              chainMasses[m - 1] -
//          kT) /
//         chainMasses[m];
//     double scale = exp(-deltaT * chainVelocities[m]);
//     chainVelocities[m] = scale * (scale * chainVelocities[m] + deltaT * G);
//   }
//
//   // put u3 here
//
//   G = (2 * kineticEnergy - ndegf * kT) / chainMasses[0];
//   for (int m = 0; m < M - 1; m++) {
//     double scale = exp(-deltaT * chainVelocities[m + 1]);
//     chainVelocities[m] = scale * (scale * chainVelocities[m] + deltaT * G);
//     G = (chainVelocities[m] * chainVelocities[m] * chainMasses[m] - kT) /
//         chainMasses[m + 1];
//   }
//   chainVelocities[M] += deltaT * G;
// }

__global__ static void coordsHalfStepVelocityUpdate(
    double kbt, const int numAtoms, const int stride, const double timeStep,
    double4 *__restrict__ coords, double4 *__restrict__ coordsDelta,
    const double4 *__restrict__ coordsDeltaPrevious,
    double4 *__restrict__ velMass, const double *__restrict__ force) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < numAtoms) {

    double fx = -force[index];
    double fy = -force[index + stride];
    double fz = -force[index + 2 * stride];

    double fact = timeStep * timeStep * velMass[index].w;

    coordsDelta[index].x = coordsDeltaPrevious[index].x + fact * fx;
    coordsDelta[index].y = coordsDeltaPrevious[index].y + fact * fy;
    coordsDelta[index].z = coordsDeltaPrevious[index].z + fact * fz;

    coords[index].x += coordsDelta[index].x;
    coords[index].y += coordsDelta[index].y;
    coords[index].z += coordsDelta[index].z;
  }
}

__global__ static void updateCoordsDeltaAfterConstraint(
    int numAtoms, const double4 *__restrict__ coordsRef,
    const double4 *__restrict__ coords, double4 *__restrict__ coordsDelta) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < numAtoms) {
    coordsDelta[index].x = coords[index].x - coordsRef[index].x;
    coordsDelta[index].y = coords[index].y - coordsRef[index].y;
    coordsDelta[index].z = coords[index].z - coordsRef[index].z;
  }
}

/** @brief Given coordsDelta of previous and next half steps, returns the
 * on-step velocity */
__global__ static void
onStepVelocityCalculation(const int numAtoms, const double timeStep,
                          double4 *__restrict__ coordsDelta,
                          double4 *__restrict__ coordsDeltaPrevious,
                          double4 *__restrict__ velMass) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < numAtoms) {

    double fact = 0.5 / timeStep;

    velMass[index].x =
        (coordsDelta[index].x + coordsDeltaPrevious[index].x) * fact;
    velMass[index].y =
        (coordsDelta[index].y + coordsDeltaPrevious[index].y) * fact;
    velMass[index].z =
        (coordsDelta[index].z + coordsDeltaPrevious[index].z) * fact;
  }
}

void CudaNoseHooverThermostatIntegrator::propagateOneStep(void) {
  auto coords = m_Context->getCoordinatesCharges().getDeviceData();
  auto xyzq = m_Context->getXYZQ()->getDeviceXYZQ();
  auto coordsDeltaDevice = m_CoordsDelta.getDeviceData();
  auto coordsDeltaPreviousDevice = m_CoordsDeltaPrevious.getDeviceData();
  auto coordsRefDevice = m_CoordsRef.getDeviceData();
  auto velMass = m_Context->getVelocityMass().getDeviceData();

  int numDegreesOfFreedom = m_Context->getDegreesOfFreedom();

  double referenceKineticEnergy =
      0.5 * numDegreesOfFreedom * charmm::constants::kBoltz * m_BathTemperature;

  if (m_DebugPrintFrequency > 0 && m_StepId % m_DebugPrintFrequency == 0) {
    std::cout << "Step id : " << m_StepId << std::endl;
  }

  int numAtoms = m_Context->getNumAtoms();
  int stride = m_Context->getForceStride();
  double kbt = charmm::constants::kBoltz * m_BathTemperature;

  if (m_StepsSinceNeighborListUpdate % m_NonbondedListUpdateFrequency == 0) {
    /*
    if (m_Context->getForceManager()->getPeriodicBoundaryCondition() ==
        PBC::P21) {
      auto groups = context->getForceManager()->getPSF()->getGroups();

      // find a better place for this
      int numGroups = groups.size();
      int numThreads = 128;
      int numBlocks = (numGroups - 1) / numThreads + 1;

      auto boxDimensions = m_Context->getBoxDimensions();
      float3 box = {(float)boxDimensions[0], (float)boxDimensions[1],
                    (float)boxDimensions[2]};

      invertDeltaAsymmetric<<<numBlocks, numThreads, 0, *m_IntegratorStream>>>(
          numGroups, groups.getDeviceData(), box.x, xyzq, stride,
          coordsDeltaPreviousDevice);
      cudaCheck(hipStreamSynchronize(*m_IntegratorStream));
    }
    */
    m_Context->resetNeighborList();
  }

  if (m_StepId % m_RemoveCenterOfMassFrequency == 0) {
    // TODO : activate this
    // removeCenterOfMassMotion();
  }

  copy_DtoD_async<double4>(coords, m_CoordsRef.getDeviceData(), numAtoms,
                           *m_IntegratorStream);

  m_Context->calculateForces(false, true, true);
  auto force = m_Context->getForces();

  m_NoseHooverPistonVelocityPrevious = m_NoseHooverPistonVelocity;
  m_NoseHooverPistonForcePrevious = m_NoseHooverPistonForce;

  int numThreads = 128;
  int numBlocks = (numAtoms - 1) / numThreads + 1;
  // int numBlocksReduction = 64;

  coordsHalfStepVelocityUpdate<<<numBlocks, numThreads, 0,
                                 *m_IntegratorStream>>>(
      kbt, numAtoms, stride, m_TimeStep, coords, coordsDeltaDevice,
      coordsDeltaPreviousDevice, velMass, force->xyz());

  cudaCheck(hipStreamSynchronize(*m_IntegratorStream));

  // TODO :  Use profiler to determine where we do this computation

  if (m_UsingHolonomicConstraints) {
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipStreamSynchronize(*m_IntegratorStream));

    m_HolonomicConstraint->handleHolonomicConstraints(coordsRefDevice);
    cudaCheck(hipDeviceSynchronize());

    // computeHolonomicConstraintForces<<<numBlocks, numThreads, 0,
    //                                    *m_IntegratorStream>>>(
    //     numAtoms, timeStep, velMass, coordsRefDevice, coords,
    //     coordsDeltaDevice,
    //     holonomicConstraintForces.getDeviceData());

    updateCoordsDeltaAfterConstraint<<<numBlocks, numThreads, 0,
                                       *m_IntegratorStream>>>(
        numAtoms, coordsRefDevice, coords, coordsDeltaDevice);

    cudaCheck(hipStreamSynchronize(*m_IntegratorStream));
    cudaCheck(hipDeviceSynchronize());
  }
  // Calculate nose hoover thermal piston velocity and position

  // TODO : change this to on step kinetic energy
  /*double onStepKineticEnergy =
      (deltaPressureHalfStepKinetic[0] + deltaPressureHalfStepKinetic[2] +
       deltaPressureHalfStepKinetic[5]) /
      (0.5 * charmm::constants::patmos / volume);
  */

  onStepVelocityCalculation<<<numBlocks, numThreads, 0, *m_IntegratorStream>>>(
      numAtoms, m_TimeStep, coordsDeltaDevice, coordsDeltaPreviousDevice,
      velMass);
  cudaCheck(hipStreamSynchronize(*m_IntegratorStream));

  double onStepKineticEnergy =
      m_Context->computeTemperature() *
      (0.5 * numDegreesOfFreedom * charmm::constants::kBoltz);
  m_NoseHooverPistonForce = 2.0 * m_TimeStep *
                            (onStepKineticEnergy - referenceKineticEnergy) /
                            m_NoseHooverPistonMass;
  if (m_NoseHooverPistonForcePrevious == 0.0) {
    m_NoseHooverPistonForcePrevious = m_NoseHooverPistonForce;
  }

  m_NoseHooverPistonVelocity =
      m_NoseHooverPistonVelocityPrevious +
      (m_NoseHooverPistonForce + m_NoseHooverPistonForcePrevious) / 2.0;

  // onStepKineticEnergy = m_Context->computeTemperature() *
  //                       (0.5 * numDegreesOfFreedom *
  //                       charmm::constants::kBoltz);
  // m_oseHooverPistonForce = 2.0 * timeStep *
  //                         (onStepKineticEnergy - referenceKineticEnergy) /
  //                         m_NoseHooverPistonMass;

  // m_NoseHooverPistonVelocity =
  //     m_NoseHooverPistonVelocityPrevious +
  //     (m_NoseHooverPistonForce + m_NoseHooverPistonForcePrevious) / 2.0;

  m_NoseHooverPistonPosition += m_NoseHooverPistonVelocity * m_TimeStep +
                                0.5 * m_NoseHooverPistonForce * m_TimeStep;

  updateSPKernel<<<numBlocks, numThreads, 0, *m_IntegratorStream>>>(
      numAtoms, xyzq, coords);

  cudaCheck(hipStreamSynchronize(*m_IntegratorStream));

  copy_DtoD_async<double4>(coordsDeltaDevice, coordsDeltaPreviousDevice,
                           numAtoms, *m_IntegratorStream);

  cudaCheck(hipStreamSynchronize(*m_IntegratorStream));

  m_Context->calculateKineticEnergy();
  auto ke = m_Context->getKineticEnergy();
  // exit if the kinetic energy is nan
  // if (ke != ke) {
  if (std::isnan(ke)) {
    throw std::runtime_error("NAN detected in kinetic energy");
    exit(1);
  }

  if (m_DebugPrintFrequency > 0 && m_StepId % m_DebugPrintFrequency == 0) {
    auto peContainer = m_Context->getPotentialEnergy();
    peContainer.transferFromDevice();
    auto pe = peContainer[0];

    std::cout << "Kinetic energy = " << ke << std::endl;

    std::cout << "Potential energy = " << pe << std::endl;
    // std::cout << "Total energy = "
    //           << pe + ke + pistonPotentialEnergy + pistonKineticEnergy +
    //           hfcten
    //           << std::endl;

    // std::cout << "HFCTE = " << hfcten << std::endl;

    std::cout << "Temperature : " << m_Context->computeTemperature() << "\n";
    std::cout << "\n";
  }

  // old code

  /*
 u4 u3 u4
 */

  // int numThreads = 128;
  // int numBlocks = (numAtoms - 1) / numThreads + 1;

  // u2Propagator<<<numBlocks, numThreads>>>(timeStep / 2.0, numAtoms, stride,
  //                                         velMass, force->xyz(), coords);

  // cudaCheck(hipDeviceSynchronize()); // TODO : remove these

  // u1Propagator<<<numBlocks, numThreads>>>(timeStep, numAtoms, velMass,
  // coords); cudaCheck(hipDeviceSynchronize()); // TODO : remove these

  // updateSPKernel<<<numBlocks, numThreads, 0, *integratorStream>>>(numAtoms,
  //                                                                 xyzq,
  //                                                                 coords);
  // cudaCheck(hipDeviceSynchronize()); // TODO : remove these
  // if (stepsSinceNeighborListUpdate % 20 == 0) {
  //   context->resetNeighborList();
  // }

  // context->calculateForces();
  // force = context->getForces();

  // u2Propagator<<<numBlocks, numThreads>>>(timeStep / 2.0, numAtoms, stride,
  //                                         velMass, force->xyz(), coords);
  // cudaCheck(hipDeviceSynchronize()); // TODO : remove these

  m_StepId++;

  return;
}
